#include "hip/hip_runtime.h"
//librerie standard
#include <iostream>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

// librerie CUDA
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

/*//librerie OPENCV
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc.hpp>*/

using namespace std;
//using namespace cv;


// Controllo errori cuda
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "\n*** FAILED - ABORTING\n"); \
            system("pause");\
            return 1; \
        } \
    } while (0)


/* Dimensioni array multidimensionale (matrici)
      N.B.: il numero di colonne della prima matrice deve coincidere assolutamente con il numero di righe della seconda matrice  */

// prima matrice (M1)
const int righeM1 = 1440;
const int colonneM1 = 2560;

// seconda matrice (M2)
const int righeM2 = 2560;
const int colonneM2 = 1440;

// dimensioni delle matrici M1, M2 e matrice dei risultati
size_t dimM1 = (righeM1*colonneM1) * sizeof(int); 
size_t dimM2 = (righeM2*colonneM2) * sizeof(int);
size_t dimRes = (righeM1*colonneM2) * sizeof(int);

/*  La matrice risultante dal prodotto avr� dimesioni righeM1 * colonneM2 (righe della prima e colonne della seonda)

     -> IL PRODOTTO TRA MATRICI NON E' COMMUTATIVO  */

// Dimensioni del blocco (x,y) impostate uguali in modo che formino blocchi quadrati esattamente di 1024 threads (limite imposto dall'hardware)
#define BLKSIZE 32


// Funzione eseguita sulla GPU (calcolo parallelo)
// viene utilizzata la memoria globale (qui le variabili infatti possono essere allocate dinamicamente con le APIs cudamalloc e cudamallocHost)
__global__ void matrix_mulGPU(int *a, int *b, int *c) {

	// Inizializzo le coordinate dei thread all'interno della griglia (col e row identificano un singolo thread specifico)
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	// Somma impostata a zero, aumenter� man mano che, come avviene nel prodotto tra matrici, si sommano gli elementi moltiplicati

	int somma = 0;

	/*  viene posta una condizione di controllo affinch� vengano chiamati solo i thread effettivamente predisposti per le operazioni di moltiplicazione
	    infatti la griglia creata � impostata sempre leggermente pi� grande delle dimensioni del prodotto righe*colonne  */

	if (row < righeM1 && col < colonneM2) {

		/*  l'iterazione parte: per la matrice M1 si percorrono le righe della griglia (thread posti orizzontalmente sull'asse x, cio� le colonneM1)
		    per la matrice M2 invece accade la stessa cosa ma si procede scorrendo lungo i thread posti verticalmente (asse y, cio� le righeM2)
		    row e col mantengono "fissato" il calcolo sulle righe e colonne corrispondenti  */

		for (int i = 0; i < colonneM1; i++) {

			// la somma accumula i prodotti che man mano vanno aggiungendosi, scorrendo infatti lungo le dimensioni x e y

			somma += a[row * colonneM1 + i] * b[i * colonneM2 + col];
			// la durata computazionale del processo � data proprio da quest'ultima stringa che dipende direttamente dalle dimensioni delle matrici in esame: il calcolo effettuato � (M1*M2)

		}
		// alla fine di ogni iterazione vengono popolati ordinatamente in modo crescente gli elementi nell'array del risultato

		c[row * colonneM2 + col] = somma;
	}
}

__global__ void matrix_mulGPUShared(int *a, int *b, int *c) {

	__shared__ int sA[BLKSIZE][BLKSIZE];   // sA usa un blocco da 32*32 = 1024 thread
	__shared__ int sB[BLKSIZE][BLKSIZE];   // sB usa un blocco da 32*32 = 1024 thread

	int row = blockDim.y * blockIdx.y + threadIdx.y;
	int col = blockDim.x * blockIdx.x + threadIdx.x;

	int somma = 0;
	sA[threadIdx.y][threadIdx.x] = 0;
	sB[threadIdx.y][threadIdx.x] = 0;

	for (int ph = 0; ph < (((colonneM1 - 1) / BLKSIZE) + 1); ph++) {
		if ((row < righeM1) && (threadIdx.x + (ph * BLKSIZE)) < colonneM1) {
			sA[threadIdx.y][threadIdx.x] = a[(row * colonneM1) + threadIdx.x + (ph * BLKSIZE)];
		}
		else {
			sA[threadIdx.y][threadIdx.x] = 0.0;
		}
		if (col < colonneM2 && (threadIdx.y + ph * BLKSIZE) < righeM2) {
			sB[threadIdx.y][threadIdx.x] = b[(threadIdx.y + ph * BLKSIZE) * colonneM2 + col];
		}
		else {
			sB[threadIdx.y][threadIdx.x] = 0.0;
		}
		__syncthreads(); // errore di Intellisense, non comporta problemi durante l'esecuzione

		for (int j = 0; j < BLKSIZE; ++j) {
			somma += sA[threadIdx.y][j] * sB[j][threadIdx.x];
		}
	}
	if (row < righeM1 && col < colonneM2) {
		c[row * colonneM2 + col] = somma;
	}
}


/*  Funzione eseguita sulla CPU (calcolo sequenziale)
    A differenza delle operazioni per cui il calcolo avviene simultaneamente per ogni fila ThreadY/ThreadX con l'unica analogia che riguarda lo scorrimento dei valori lungo le fasce,
	la CPU, operando in modo sequenziale, deve scorrerere un elemento per volta e moltiplicarlo per il giusto elemento dell'altra matrice  */

void matrix_mulCPU(int* a, int* b, int* c) {

	for (int i = 0; i < righeM1; i++) {
		for (int j = 0; j < colonneM2; j++) {
			//ogni volta avviene un reset della somma
			 int somma = 0;

			for (int k = 0; k < colonneM1; k++) {
				somma += a[i * colonneM1 + k] * b[k * colonneM2 + j];
			}

			c[i * colonneM2 + j] = somma;
		}
	}

	return;
}


int main() {

	/*       VARIANTE CON IMMAGINI

	//apro l'immagine e la carico nella variabile img
	// la conversione in scala di grigi serve per semplificare e avere matrici in due dimensioni invece  di tre
	puts("Acquisizione dati immagine");
	Mat imgOriginal = imread("image.jpg", IMREAD_GRAYSCALE);

	// controllo che siano presenti i dati dell'immagine
	if (imgOriginal.data == NULL) {
		cerr << "Errore nell'aprire l'immagine" << endl;
		return(-1);
	}

	// adattamento dell' immagine alle dimensioni della matrice

	Mat imgResized;
	double scale_x = colonneM1 / (int)imgOriginal.cols;
	double scale_y = righeM1 / (int)imgOriginal.rows;
	resize(imgOriginal, imgResized, Size(), scale_x, scale_y, INTER_LINEAR);
	Mat img = imgResized;

	puts("Acquisizione completata");
	cout << endl;

	Si va poi a popolare la matrice host con i dati dei pixel dell'immagine iterando il comando matriceHost[i*colonneM1 + j]=(int)img.at<uchar>(i,j)
	I valori per un canale solo (Greyscale) vanno da 0 a 255

	*/

	hipFree(0);

	// Creazione Cuda event, servir� per calcolare la durata delle operazioni che riguardano la GPU
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	float elapsed1 = 0; // time in ms
	float elapsed2 = 0; // time in ms
	float elapsed3 = 0; // time in ms

	// Clock per calcolare la durata della funzione eseguita sulla CPU
	clock_t inizio, fine;
	float tempo;

	puts(" ____Operazioni di moltiplicazione matriciale a conftonto CPU vs GPU____");
	cout << endl;

	// Restituisce alcuni parametri della scheda NVidia in uso
	hipDeviceProp_t prop;
	size_t free1, free2, total;

	hipGetDeviceProperties(&prop, 0);
		cudaCheckErrors("Errore acquisizione dati");
	hipMemGetInfo(&free1, &total);
		cudaCheckErrors("Errore acquisizione dati");

	printf(" - Device: %s\n", prop.name);
	cout << " - GPU -> memory: free = " << free1 / (1024 * 1024) << " MegaBytes, total = " << total / (1024*1024*1024) << " GigaBytes" << endl;
	cout << endl;

	// Condizione di controllo  per confrontare le dimensioni delle matrici e non generare eccezioni durante l'elaborazione
	if (righeM2 != colonneM1) {

		cout << "Le colonne della prima matrice non corrispondono alle righe della seconda" << endl;
		cout << "ESECUZIONE ARRESTATA, ATTENZIONE IMMETTERE VALORI UGUALI PER LE DUE DIMENSIONI" << endl << endl;

		system("pause");
		return -1;
	}


	puts(" -- Allocazione delle variabili Host nella memoria --");
	// Allocazione matrice che si andr� a moltiplicare a quelle presenti nelle memorie

	int* matRandHost;
	hipHostMalloc((void **)&matRandHost, dimM2);

	// Generazione valori randomici e popolamento matrice
	puts(" - Generazione valori randomici per la prima e la seconda matrice");

	for (int i = 0; i < righeM2; i++)
		for (int j = 0; j < colonneM2; j++)
			matRandHost[i*colonneM2 + j] = rand() % 256;

	// Allocazione matrice di host
	int* matriceHost;
	hipHostMalloc((void **)&matriceHost, dimM1);

	// Popolamento matrice con valori randomici
	for (int i = 0; i < righeM1; i++)
		for (int j = 0; j < colonneM1; j++)
			matriceHost[i*colonneM1 + j] = rand() % 256;
	//matriceHost[i*colonneM1 + j] = (int)img.at<uchar>(i, j);

	// Matrici dei risultati e relativa allocazione
	int *matResHost, *matResHostSH, *matResCPU;

	hipHostMalloc((void **)&matResHost, dimRes);
	hipHostMalloc((void **)&matResHostSH, dimRes);
	matResCPU = (int *)malloc(dimRes);

	puts(" - Allocazione e popolamento matrici completati");
		cout << endl;


	// Allocazione di memoria per le variabili che lavoreranno sulla GPU
	puts(" -- Allocazione variabili nella memoria della GPU --");

	int *matriceGPU, *matRGPU, *matResGPU, *matResGPUSH;

	hipMalloc((void **)&matriceGPU, dimM1);
		cudaCheckErrors("Allocazione fallita");
	hipMalloc((void **)&matRGPU, dimM2);
		cudaCheckErrors("Allocazione fallita");
	hipMalloc((void **)&matResGPU, dimRes);
		cudaCheckErrors("Allocazione fallita");
	hipMalloc((void **)&matResGPUSH, dimRes);
		cudaCheckErrors("Allocazione fallita");

	puts(" - Allocazione completata");
		cout << endl;


	// Copia dei valori della prima e seconda matrice (host) nelle variabili device
	puts(" -- Trasferimento valori delle due matrici nella GPU --");

	hipEventRecord(start);

	hipMemcpy(matriceGPU, matriceHost, dimM1, hipMemcpyHostToDevice);
		cudaCheckErrors("Copia dei dati da Host a Device fallita");
	hipMemcpy(matRGPU, matRandHost, dimM2, hipMemcpyHostToDevice);
		cudaCheckErrors("Copia dei dati da Host a Device fallita");
	hipMemcpy(matResGPU, matResHost, dimRes, hipMemcpyHostToDevice);
		cudaCheckErrors("Copia dei dati da Host a Device fallita");

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed1, start, stop);

	puts(" - Trasferimento completato");
		cout << " - Tempo trascorso: " << elapsed1 << " ms" << endl;
	hipMemGetInfo(&free2, &total);
	cout << " - GPU -> memory: memoria occupata dalle matrici = " << (free1 - free2) / (1024*1024) << " MegaBytes" << endl;
		cout << endl;


	// Dimensionamento della griglia di blocchi e thread (max 1024 thread per blocco)
	puts(" -- Costruzione griglia di calcolo per la GPU --");

	dim3 block(BLKSIZE, BLKSIZE); // 32 * 32 = 1024 (colonne,righe)
	dim3 grid((int)ceil((colonneM2 + BLKSIZE - 1) / BLKSIZE), (int)ceil((righeM1 + BLKSIZE - 1) / BLKSIZE)); //trovo il valore intero pi� grande per costruire la griglia di dimensioni adeguate (colonne,righe)

	cout << endl;


	// Esecuzione funzione sulla GPU
	puts(" -- Avvio calcolo sulla GPU --");

	hipEventRecord(start);

	matrix_mulGPU << <grid, block >> > (matriceGPU, matRGPU, matResGPU);
		cudaCheckErrors("Esecuzione del kernel Fallita");

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed2, start, stop);


	// Condizione del WDDM TDR DELAY
	if ((elapsed2 / 1000) == 0 || (elapsed2 / 1000) > 5) {
		cout << endl << "Sono passati piu' di 5 secondi, e' intervenuto il  TIMEOUT DETECTION & RECOVERY (WDDM TDR DELAY)" << endl;
		cout << "PROGRAMMA ARRESTATO, IMMETTERE MATRICI DI DIMENSIONI MINORI" << endl << endl;
		system("pause");
			return -1;
	}

	puts(" - Calcolo sulla GPU completato");
	cout << " - Tempo trascorso: " << elapsed2 / 1000 << " s" << endl;
	cout << endl;


	// Esecuzione funzione sulla GPU usando la shared memory
	puts(" -- Avvio calcolo sulla GPU utilizzando la Shared Memory --");

	hipEventRecord(start);

	matrix_mulGPUShared << <grid, block >> > (matriceGPU, matRGPU, matResGPUSH);
	cudaCheckErrors("Esecuzione del kernel Fallita");

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed2, start, stop);


	// Condizione del WDDM TDR DELAY
	if ((elapsed2 / 1000) == 0 || (elapsed2 / 1000) > 5) {
		cout << endl << "Sono passati piu' di 5 secondi, e' intervenuto il  TIMEOUT DETECTION & RECOVERY (WDDM TDR DELAY)" << endl;
		cout << "PROGRAMMA ARRESTATO, IMMETTERE MATRICI DI DIMENSIONI MINORI" << endl << endl;
		system("pause");
		return -1;
	}

	puts(" - Calcolo sulla GPU completato");
	cout << " - Tempo trascorso: " << elapsed2 / 1000 << " s" << endl;
	cout << endl;


	// Trasferimento dei valori della matrice risultante dalla compilazione sulla GPU alla variabile Host
	puts(" -- Trasferimento valori della GPU alla matrice del Host del risultato --");

	hipEventRecord(start);

	hipMemcpyAsync(matResHost, matResGPU, dimRes, hipMemcpyDeviceToHost);
		cudaCheckErrors("Trasferimento fallito\n");

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed3, start, stop);

	puts(" - Trasferimento completato");
	cout << " - Tempo trascorso: " << elapsed3 << " ms" << endl;
	printf(" - Larghezza di banda utilizzata (Device2H) per trasferire la matrice dei risultati (GB/s): %f\n", dimRes * 1e-6 / elapsed3);
		cout << endl;


		// Trasferimento dei valori della matrice risultante dalla compilazione sulla GPU alla variabile Host (SHARED MEMORY)
		puts(" -- Trasferimento valori della GPU alla matrice del Host del risultato (SM) --");

		hipEventRecord(start);

		hipMemcpyAsync(matResHostSH, matResGPUSH, dimRes, hipMemcpyDeviceToHost);
		cudaCheckErrors("Trasferimento fallito\n");

		hipEventRecord(stop);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed3, start, stop);

		puts(" - Trasferimento completato");
		cout << " - Tempo trascorso: " << elapsed3 << " ms" << endl;
		printf(" - Larghezza di banda utilizzata (Device2H) per trasferire la matrice dei risultati (GB/s): %f\n", dimRes * 1e-6 / elapsed3);
		cout << endl;


	// Esecuzione funzione sulla CPU
	puts(" -- Avvio calcolo sulla CPU --");

	inizio = clock();

	matrix_mulCPU(matriceHost, matRandHost, matResCPU);

	fine = clock();
	tempo = ((float)(fine - inizio)) / CLOCKS_PER_SEC;

	puts(" - Calcolo sulla CPU eseguito");
	cout << " - Tempo trascorso: " << tempo << " s" << endl;
		cout << endl;


	// Funzione di confronto degli elementi nelle matrici ottenute dalla CPU e dalla GPU
	puts(" -- Controllo dei risultati --");
	bool esito = true;

	for (int i = 0; i < righeM1; i++) {
		if (esito != false) {
			for (int j = 0; j < colonneM2; j++) {
				if (matResCPU[i*colonneM2 + j] != matResHost[i*colonneM2 + j]) {
					cout << " --> ERRORE" << endl << endl;
					esito = false;
					break;
				}
				else if (matResCPU[i*colonneM2 + j] != matResHostSH[i*colonneM2 + j]) {
					cout << " --> ERRORE" << endl << endl;
					esito = false;
					break;
				}
				else if (matResHostSH[i*colonneM2 + j] != matResHost[i*colonneM2 + j]) {
					cout << " --> ERRORE" << endl << endl;
					esito = false;
					break;
				}
			}
		}
		else
			break;
	}

	if (esito)
		puts(" - Esito: completato senza aver riscontrato errori");
	else
		cout << " - Esito: ATTENZIONE SONO STATI RILEVATI VALORI DISCORDANTI";


	hipFree(matriceGPU);
	hipFree(matRGPU);
	hipFree(matResGPU);
	hipFree(matResGPUSH);

	hipHostFree(matRandHost);
	hipHostFree(matriceHost);
	hipHostFree(matResHost);
	hipHostFree(matResHostSH);

	cout << endl << endl;
	system("pause");

		return 0;
}