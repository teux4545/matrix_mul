#include "hip/hip_runtime.h"
// librerie CUDA
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

//librerie standard
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

//librerie OPENCV
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc.hpp>

using namespace std;
using namespace cv;


// Controllo errori cuda
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "\n*** FAILED - ABORTING\n"); \
            system("pause");\
            return 1; \
        } \
    } while (0)

hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA Runtime Error: %s\n",
			hipGetErrorString(result));
		assert(result == hipSuccess);
	}
#endif
	return result;
}


// Dimensioni array multidimensionale (matrici)
// NB il numero di colonne della prima matrice deve coincidere assolutamente con il numero di righe della seconda matrice
// oppure, viceversa, le righe della prima dovranno coincidere con le colonne della seconda

//prima matrice (M1)
const int righeM1 = 1500;
const int colonneM1 = 1500;

//seconda matrice (M2)
const int righeM2 = 1500;
const int colonneM2 = 1500;

// la matrice risultante dal prodotto avr� dimesioni (colonneM1 * righeM2) o (righeM1 * colonneM2) 
// a seconda se facciamo rispettivamente M2*M1 oppure M1*M2

#define BLKSIZE 32


// funzione eseguita sulla GPU (calcolo parallelo)
__global__ void matrix_mulGPU(int *a, int *b, int *c){

	// Compute each thread's global row and column index
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	int sum = 0;
	//col<righe si riferisce alle colonne della seconda matrice che infatti nel nostro caso sono le righe
	if (row < righeM1 && col < colonneM2) {
		
		for (int i = 0; i < colonneM1; i++) {

			sum += (a[row * colonneM1 + i] * b[i * colonneM2 + col])/255;
		}
		c[row * colonneM2 + col] = sum;
	}
}

void matrix_mulCPU(int* a, int* b, int* c)
{
	int somma = 0;
	//righe della prima matrice
	for (int i = 0; i < righeM1; i++)
	{
		//colonne della seconda matrice che corrispondono infatti alle righe della prima matrice
		//(definizione di prodotto tra matrici)
		for (int j = 0; j < colonneM2; j++)
		{
			somma = 0;
			//colonne della prima matrice
			for (int k = 0; k < colonneM1; k++)
			{
				//colonne prima matrice -> colonne , colonne seconda matrice - > righe
				somma += (a[i * colonneM1 + k] * b[k * colonneM2 + j])/255;
			}
			c[i * colonneM2 + j] = somma;
			
		}
	}
}


int main() {

	/*       VARIANTE CON IMMAGINI
	
	//apro l'immagine e la carico nella variabile img
	// la conversione in scala di grigi serve per semplificare e avere matrici in due dimensioni invece  di tre
	puts("Acquisizione dati immagine");
	Mat imgOriginal = imread("image.jpg", IMREAD_GRAYSCALE);
	
	// controllo che siano presenti i dati dell'immagine
	if (imgOriginal.data == NULL) {
		cerr << "Errore nell'aprire l'immagine" << endl;
		return(-1);
	}

	// adattamento dell' immagine alle dimensioni della matrice

	Mat imgResized;
	double scale_x = colonneM1 / (int)imgOriginal.cols;
	double scale_y = righeM1 / (int)imgOriginal.rows;
	resize(imgOriginal, imgResized, Size(), scale_x, scale_y, INTER_LINEAR);
	Mat img = imgResized;

	puts("Acquisizione completata");
	cout << endl;
	*/
	puts("Allocazione delle variabili Host nella memoria");
	// allocazione  matrice che si andr� a moltiplicare a quelle presenti nelle memorie

	int* matRandHost;
	matRandHost = (int *)malloc((colonneM2*righeM2) * sizeof(int));

	// generazione valori pseudorandomici e popolamento matrice
	puts("Generazione valori randomici per la prima e la seconda matrice");

	for (int i = 0; i < righeM2; i++)
		for (int j = 0; j < colonneM2; j++)
			matRandHost[i*colonneM2 + j] = rand() % 256;

	// allocazione matrice di host
	int* matriceHost;
	matriceHost = (int *)malloc(righeM1*colonneM1 * sizeof(int));

	// popolamento matrice con i valori dell' immagine
	for (int i = 0; i < righeM1; i++)
		for (int j = 0; j < colonneM1; j++)
			matriceHost[i*colonneM1 + j] = rand() % 256;
			//matriceHost[i*colonneM1 + j] = (int)img.at<uchar>(i, j);

	int *matResHost, *matResCPU;
	matResHost = (int *)malloc(righeM1*colonneM2 * sizeof(int));
	matResCPU = (int *)malloc(righeM1*colonneM2 * sizeof(int));

	puts("Allocazione popolamento matrici completati");
	cout << endl;

	// Allocazione di memoria per le variabili che lavoreranno sulla GPU
	puts("Allocazione variabili nella memoria della GPU");

	int *matriceGPU, *matRGPU, *matResGPU;

	hipMalloc((int **)&matriceGPU, (righeM1*colonneM1) * sizeof(int));
	cudaCheckErrors("Allocazione fallita");
	hipMalloc((int **)&matRGPU, (righeM2*colonneM2) * sizeof(int));
	cudaCheckErrors("Allocazione fallita");
	hipMalloc((int **)&matResGPU, (righeM1*colonneM2) * sizeof(int));
	cudaCheckErrors("Allocazione fallita");

	puts("Allocazione completata");
	cout << endl ;

	//Copia dei valori della prima e seconda matrice (host) nelle variabili device
	puts("Trasferimento valori delle due matrici nella GPU");

	hipMemcpy(matriceGPU, matriceHost, (righeM1*colonneM1) * sizeof(int), hipMemcpyHostToDevice);
	cudaCheckErrors("Copia dei dati da Host a Device fallita");
	hipMemcpy(matRGPU, matRandHost, (righeM2*colonneM2) * sizeof(int), hipMemcpyHostToDevice);
	cudaCheckErrors("Copia dei dati da Host a Device fallita");
	hipMemcpy(matResGPU, matResHost, (righeM1*colonneM2) * sizeof(int), hipMemcpyHostToDevice);
	cudaCheckErrors("Copia dei dati da Host a Device fallita");

	puts("Trasferimento completato");
	cout << endl;

	// Dimensionamento della griglia di blocchi e thread (max 1024 thread per blocco)
	puts("Costruzione griglia di calcolo per la GPU");

	dim3 threads(BLKSIZE, BLKSIZE);
	dim3 blocks((colonneM2 + BLKSIZE - 1)/ BLKSIZE, (righeM1 * BLKSIZE - 1 )/ BLKSIZE);

	cout << endl;

	// Esecuzione funzione sulla GPU
	puts("Avvio calcolo sulla GPU");

	matrix_mulGPU << <blocks, threads >> > (matriceGPU, matRGPU, matResGPU);
	cudaCheckErrors("Esecuzione del kernel Fallita");
	hipDeviceSynchronize();
	puts("Calcolo sulla GPU eseguito");
	cout << endl;

	//Trasferimento dei valori della matrice risultante dalla compilazione sulla GPU alla variabile Host
	puts("Trasferimento valori della GPU alla matrice del Host del risultato");
	hipMemcpyAsync(matResHost, matResGPU, (righeM1*colonneM2) * sizeof(int), hipMemcpyDeviceToHost);
	cudaCheckErrors("Copia dei dati fallita\n");
	puts("Trasferimento completato");
	cout << endl;

	// Esecuzione funzione sulla CPU
	puts("Avvio calcolo sulla CPU");
	matrix_mulCPU(matriceHost, matRandHost, matResCPU);
	puts("Calcolo sulla CPU eseguito");
	cout << endl << endl;

		free(matRandHost);
		free(matriceHost);
		free(matResHost);

		hipFree(matriceGPU);
		hipFree(matRGPU);
		hipFree(matResGPU);

		system("pause");
	return 0;
}